#include "hip/hip_runtime.h"
/*
** xkaapi
** 
** Created on Tue Mar 31 15:19:14 2009
** Copyright 2009 INRIA.
**
** Contributors :
** thierry.gautier@inrialpes.fr
** fabien.lementec@imag.fr
** 
** This software is a computer program whose purpose is to execute
** multithreaded computation with data flow synchronization between
** threads.
** 
** This software is governed by the CeCILL-C license under French law
** and abiding by the rules of distribution of free software.  You can
** use, modify and/ or redistribute the software under the terms of
** the CeCILL-C license as circulated by CEA, CNRS and INRIA at the
** following URL "http://www.cecill.info".
** 
** As a counterpart to the access to the source code and rights to
** copy, modify and redistribute granted by the license, users are
** provided only with a limited warranty and the software's author,
** the holder of the economic rights, and the successive licensors
** have only limited liability.
** 
** In this respect, the user's attention is drawn to the risks
** associated with loading, using, modifying and/or developing or
** reproducing the software by the user in light of its specific
** status of free software, that may mean that it is complicated to
** manipulate, and that also therefore means that it is reserved for
** developers and experienced professionals having in-depth computer
** knowledge. Users are therefore encouraged to load and test the
** software's suitability as regards their requirements in conditions
** enabling the security of their systems and/or data to be ensured
** and, more generally, to use and operate it in the same conditions
** as regards security.
** 
** The fact that you are presently reading this means that you have
** had knowledge of the CeCILL-C license and that you accept its
** terms.
** 
*/

// this is a testing code for 2d ranges. it computes a scaled
// (averaged) output from a source image. it should not be
// used for benchmarking, since the kernel does not compute
// enough, and memory transfers are to small.


#include "kaapi++"
#include <algorithm>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdint.h>
#include <sys/types.h>


// static configuration
// output must be square root of input dim
// otherwise the following code does not work 
#if 0
#define CONFIG_INPUT_DIM 1024
#define CONFIG_OUTPUT_DIM 32
#else
#define CONFIG_INPUT_DIM 4096
#define CONFIG_OUTPUT_DIM 64
#endif

// split is done on an output row basis
// otherwise, split done on a cell basis.
// non row mode should be used to tests 2d
// copy memory transfers since it bypass
// the contiguous optimization case.
#define CONFIG_USE_ROW 0

// compare with seq result
#define CONFIG_USE_CHECK 0

// time executions
#define CONFIG_USE_TIME 1


// image helpers

typedef uint64_t pixel_type;

static void create_images
(pixel_type*& in, pixel_type*& out_par, pixel_type*& out_seq)
{
  const size_t isize =
    CONFIG_INPUT_DIM * CONFIG_INPUT_DIM * sizeof(pixel_type);

  const size_t osize =
    CONFIG_OUTPUT_DIM * CONFIG_OUTPUT_DIM * sizeof(pixel_type);

  in = (pixel_type*)malloc(isize);

  for (size_t i = 0; i < (isize / sizeof(pixel_type)); ++i)
    in[i] = i & 0xff;

  out_par = (pixel_type*)malloc(osize);
  out_seq = (pixel_type*)malloc(osize);
}

static void destroy_images
(pixel_type* in, pixel_type* out_par, pixel_type* out_seq)
{
  free((void*)in);
  free((void*)out_par);
  free((void*)out_seq);
}

#if CONFIG_USE_CHECK
static int compare_images
(const pixel_type* a, const pixel_type* b)
{
  const pixel_type* const saved = a;

  size_t size = CONFIG_OUTPUT_DIM * CONFIG_OUTPUT_DIM;
  for (; size; --size, ++a, ++b)
  {
    if (!(*a == *b))
    {
      printf("invalid at %lx,%lu,%lu: %lu != %lu\n",
	     (uintptr_t)a,
	     (a - saved) / CONFIG_OUTPUT_DIM,
	     (a - saved) % CONFIG_OUTPUT_DIM,
	     *a, *b);
      return -1;
    }
  }
  return 0;
}
#endif // CONFIG_USE_CHECK

static pixel_type sum_block_pixels(const pixel_type* in)
{
  static const size_t w = CONFIG_OUTPUT_DIM;

  // substract w since adavanced in previous iteration
  static const size_t lda = CONFIG_INPUT_DIM - CONFIG_OUTPUT_DIM;

  pixel_type sum = 0;

  for (size_t i = 0; i < w; ++i, in += lda)
    for (size_t j = 0; j < w; ++j, ++in)
      sum += *in;

  return sum;
}

static void scale_image_seq(pixel_type* in, pixel_type* out)
{
  static const size_t stride = (CONFIG_OUTPUT_DIM - 1) * CONFIG_INPUT_DIM;

  static const size_t w = CONFIG_OUTPUT_DIM;
  static const pixel_type ww = CONFIG_OUTPUT_DIM * CONFIG_OUTPUT_DIM;

  // out[i] = sum(inblocks[i]) / ww;
  for (size_t i = 0; i < w; ++i, in += stride)
    for (size_t j = 0; j < w; ++j, in += w, ++out)
      *out = sum_block_pixels(in) / ww;
}

// scale parallel version

class ScaleWork
{
private:
  pixel_type* _in;
  pixel_type* _out;
  kaapi_workqueue_t _wq;

public:
  ScaleWork(pixel_type* in, pixel_type* out)
    : _in(in), _out(out)
  {
#if CONFIG_USE_ROW
    // output row count
    const size_t orows = CONFIG_OUTPUT_DIM;
    kaapi_workqueue_init(&_wq, 0, orows);
#else
    kaapi_workqueue_init
      (&_wq, 0, CONFIG_OUTPUT_DIM * CONFIG_OUTPUT_DIM);
#endif
  }

  bool extractPar(size_t& unit)
  {
    kaapi_workqueue_index_t i, j;
    if (kaapi_workqueue_steal(&_wq, &i, &j, 1)) return false;
    unit = i;
    return true;
  }

  bool extractSeq(size_t& unit)
  {
    kaapi_workqueue_index_t i, j;
    if (kaapi_workqueue_pop(&_wq, &i, &j, 1)) return false;
    unit = i;
    return true;
  }

  ka::array<2, pixel_type> unit_to_iarr(size_t unit)
  {
#if CONFIG_USE_ROW
    // unit an OUTPUT row
    pixel_type* const p = _in + unit * CONFIG_INPUT_DIM * CONFIG_OUTPUT_DIM;
    return ka::array<2, pixel_type>
      (p, CONFIG_OUTPUT_DIM, CONFIG_INPUT_DIM, CONFIG_INPUT_DIM);
#else
    // unit an OUTPUT cell

    const size_t row = unit / CONFIG_OUTPUT_DIM;
    const size_t col = unit % CONFIG_OUTPUT_DIM;

    pixel_type* const p =
      _in + (row * CONFIG_INPUT_DIM + col) * CONFIG_OUTPUT_DIM;

    return ka::array<2, pixel_type>
      (p, CONFIG_OUTPUT_DIM, CONFIG_OUTPUT_DIM, CONFIG_INPUT_DIM);
#endif
  }

  ka::array<1, pixel_type> unit_to_oarr(size_t unit)
  {
#if CONFIG_USE_ROW
    const size_t col_count = CONFIG_OUTPUT_DIM;
#else
    const size_t col_count = 1;
#endif
    pixel_type* const p = _out + unit * col_count;
    return ka::array<1, pixel_type>(p, col_count);
  }

  void split(ka::StealContext*, int, ka::Request*);
};

struct ScaleTask : public ka::Task<2>::Signature
<
  ka::R<ka::range2d<pixel_type> >,
  ka::W<ka::range1d<pixel_type> >
>{};

template<>
struct TaskBodyCPU<ScaleTask>
{
  void operator()
  (
   ka::range2d_r<pixel_type> in,
   ka::range1d_w<pixel_type> out
  )
  {
    // not implemented, assume gpu only
  }
};


#if CONFIG_USE_ROW

__global__ void ScaleKernelCuda
(const pixel_type* in, pixel_type* out)
{
  // out the output row to compute
  // inthe 2d matrix

  __shared__ pixel_type shared_sums[CONFIG_OUTPUT_DIM];

  // each cuda block works on it own input
  in = in + blockIdx.x * CONFIG_OUTPUT_DIM + threadIdx.x;

  // each thread sum its column in shared_sum[x]
  pixel_type local_sum = 0;
  for (size_t i = 0; i < CONFIG_OUTPUT_DIM; ++i, in += CONFIG_INPUT_DIM)
    local_sum += *in;
  shared_sums[threadIdx.x] = local_sum;

  syncthreads();

  if (threadIdx.x == 0)
  {
    const pixel_type ww = CONFIG_OUTPUT_DIM * CONFIG_OUTPUT_DIM;

    // reduce sums in out[blockDim.x]
    local_sum = 0;
    for (size_t i = 0; i < CONFIG_OUTPUT_DIM; ++i)
      local_sum += shared_sums[i];
    out[blockIdx.x] = local_sum / ww;
  }
}

static void ScaleKernelCpu
(const pixel_type* in, pixel_type* out)
{
  static const size_t w = CONFIG_OUTPUT_DIM;
  static const pixel_type ww = CONFIG_OUTPUT_DIM * CONFIG_OUTPUT_DIM;

  for (size_t i = 0; i < w; ++i, ++out, in += w)
    *out = sum_block_pixels(in) / ww;
}

#else // ! CONFIG_USE_ROW

__global__ void ScaleKernelCuda
(const pixel_type* in, pixel_type* out)
{
  // out the output row to compute
  // inthe 2d matrix

  __shared__ pixel_type shared_sums[CONFIG_OUTPUT_DIM];

  // each cuda block works on it own input
  in = in + threadIdx.x;

  // each thread sum its column in shared_sum[x]
  pixel_type local_sum = 0;
  for (size_t i = 0; i < CONFIG_OUTPUT_DIM; ++i, in += CONFIG_OUTPUT_DIM)
    local_sum += *in;
  shared_sums[threadIdx.x] = local_sum;

  syncthreads();

  if (threadIdx.x == 0)
  {
    const pixel_type ww = CONFIG_OUTPUT_DIM * CONFIG_OUTPUT_DIM;

    local_sum = 0;
    for (size_t i = 0; i < CONFIG_OUTPUT_DIM; ++i)
      local_sum += shared_sums[i];

    *out = local_sum / ww;
  }
}

static void ScaleKernelCpu
(const pixel_type* in, pixel_type* out)
{
  *out = sum_block_pixels(in) / (CONFIG_OUTPUT_DIM * CONFIG_OUTPUT_DIM);
}

#endif // CONFIG_USE_ROW

template<>
struct TaskBodyGPU<ScaleTask>
{
  void operator()
  (
   ka::gpuStream stream,
   ka::range2d_r<pixel_type> in,
   ka::range1d_w<pixel_type> out
  )
  {
    // 1 input block per SM
    // 1 thread per input block col

#if CONFIG_USE_ROW
    static const size_t block_count = CONFIG_OUTPUT_DIM;
#else
    static const size_t block_count = 1;
#endif

    const hipStream_t custream = (hipStream_t)stream.stream;
    ScaleKernelCuda<<<block_count, CONFIG_OUTPUT_DIM, 0, custream>>>
      (in.ptr(), out.ptr());
  }
};


void ScaleWork::split(ka::StealContext* sc, int nreq, ka::Request* req)
{
  size_t unit;

  for (; nreq; --nreq, ++req)
  {
    if (extractPar(unit) == false) return ;

    ka::array<2, pixel_type> iarr = unit_to_iarr(unit);
    ka::array<1, pixel_type> oarr = unit_to_oarr(unit);

    req->Spawn<ScaleTask>(sc)(iarr, oarr);
  }
}


static void scale_image_par(pixel_type* in, pixel_type* out)
{
  ka::StealContext* sc;

  ScaleWork work(in, out);

  sc = ka::TaskBeginAdaptive
  (
   KAAPI_SC_CONCURRENT | KAAPI_SC_NOPREEMPTION,
   &ka::WrapperSplitter<ScaleWork, &ScaleWork::split>,
   &work
  );

  size_t unit;

  while (work.extractSeq(unit))
  {
    ka::array<2, pixel_type> iarr = work.unit_to_iarr(unit);
    ka::array<1, pixel_type> oarr = work.unit_to_oarr(unit);
    ScaleKernelCpu(iarr.ptr(), oarr.ptr());
  }

  ka::TaskEndAdaptive(sc);
}


// main task
struct doit
{
  void operator()(int argc, char** argv )
  {
    pixel_type* in;
    pixel_type* out_seq;
    pixel_type* out_par;

#if CONFIG_USE_TIME
    double t0, t1;
#endif

    create_images(in, out_par, out_seq);

#if CONFIG_USE_TIME
    t0 = kaapi_get_elapsedtime();
#endif

    scale_image_seq(in, out_seq);

#if CONFIG_USE_TIME
    t1 = kaapi_get_elapsedtime();
    std::cout << t1 - t0 << std::endl; // seconds
#endif

    for (size_t iter = 0; iter < 10; ++iter)
    {
#if CONFIG_USE_TIME
      t0 = kaapi_get_elapsedtime();
#endif

      scale_image_par(in, out_par);

#if CONFIG_USE_TIME
      t1 = kaapi_get_elapsedtime();
      std::cout << t1 - t0 << std::endl; // seconds
#endif

#if CONFIG_USE_CHECK
      if (compare_images(out_par, out_seq))
	printf("invalid\n");
#endif
    }

    destroy_images(in, out_par, out_seq);
  }
};


int main(int argc, char** argv)
{
  try
  {
    ka::Community com = ka::System::join_community(argc, argv);
    ka::SpawnMain<doit>()(argc, argv); 
    com.leave();
    ka::System::terminate();
  }
  catch (const std::exception& E)
  {
    ka::logfile() << "Catch : " << E.what() << std::endl;
  }
  catch (...)
  {
    ka::logfile() << "Catch unknown exception: " << std::endl;
  }

  return 0;
}
