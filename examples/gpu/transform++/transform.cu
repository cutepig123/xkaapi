#include "hip/hip_runtime.h"
/*
** xkaapi
** 
** Created on Tue Mar 31 15:19:14 2009
** Copyright 2009,2010,2011,2012 INRIA.
**
** Contributors :
** thierry.gautier@inrialpes.fr
** fabien.lementec@imag.fr
** 
** This software is a computer program whose purpose is to execute
** multithreaded computation with data flow synchronization between
** threads.
** 
** This software is governed by the CeCILL-C license under French law
** and abiding by the rules of distribution of free software.  You can
** use, modify and/ or redistribute the software under the terms of
** the CeCILL-C license as circulated by CEA, CNRS and INRIA at the
** following URL "http://www.cecill.info".
** 
** As a counterpart to the access to the source code and rights to
** copy, modify and redistribute granted by the license, users are
** provided only with a limited warranty and the software's author,
** the holder of the economic rights, and the successive licensors
** have only limited liability.
** 
** In this respect, the user's attention is drawn to the risks
** associated with loading, using, modifying and/or developing or
** reproducing the software by the user in light of its specific
** status of free software, that may mean that it is complicated to
** manipulate, and that also therefore means that it is reserved for
** developers and experienced professionals having in-depth computer
** knowledge. Users are therefore encouraged to load and test the
** software's suitability as regards their requirements in conditions
** enabling the security of their systems and/or data to be ensured
** and, more generally, to use and operate it in the same conditions
** as regards security.
** 
** The fact that you are presently reading this means that you have
** had knowledge of the CeCILL-C license and that you accept its
** terms.
** 
*/
#include <stdint.h>
#include <stdlib.h>
#include <sys/types.h>
#include <hip/hip_runtime.h>
#include "kaapi++.h"
#include "for_each_work.h"

// missing decls
typedef uintptr_t kaapi_mem_addr_t;
extern "C" void kaapi_mem_delete_host_mappings(kaapi_mem_addr_t, size_t);
typedef float double_type;

__global__ void add1(double_type* array, unsigned int size)
{
  const unsigned int per_thread = size / blockDim.x;
  unsigned int i = threadIdx.x * per_thread;

  unsigned int j = size;
  if (threadIdx.x != (blockDim.x - 1)) j = i + per_thread;

  for (; i < j; ++i) ++array[i];
}

/** Simple thief task that only do sequential computation
*/
template<typename T, typename OP>
struct TaskBodyCPU<TaskThief<T, OP> > {
  void operator()(ka::range1d_rw<T> range, OP op)
  {
    printf("cpuTask(0x%lx, %lu)\n",
	   (uintptr_t)range.begin(), range.size());

    T* const beg = range.begin();
    T* const end = beg + range.size();
    std::for_each( beg, end, op );
  }
};

extern "C" hipStream_t kaapi_cuda_kernel_stream(void);

template<typename T, typename OP>
struct TaskBodyGPU<TaskThief<T, OP> > {
  void operator()
  (ka::gpuStream stream, ka::range1d_rw<T> range, OP)
  {
    const hipStream_t custream = (hipStream_t)stream.stream;

    printf("cudaTask(0x%lx 0x%lx, %lu)\n",
	   (uintptr_t)custream, (uintptr_t)range.begin(), range.size());

    add1<<<1, 256, 0, custream>>>(range.begin(), range.size());
  }

  void operator()(ka::range1d_rw<T> range, OP op)
  {
    // helper to bypass a bug in code generation
    ka::gpuStream gpustream
      ((kaapi_gpustream_t)kaapi_cuda_kernel_stream());
    (*this)(gpustream, range, op);
  }
};

/* For each main function */
template<typename T, class OP>
static void for_each( T* beg, T* end, OP op )
{
  T* const base = beg;
  const size_t size = (size_t)(end - beg) * sizeof(T*);

  /* range to process */
  ka::StealContext* sc;
  Work<T,OP> work(beg, end, op);

  /* push an adaptive task */
  const int flags = KAAPI_SC_CONCURRENT | KAAPI_SC_NOPREEMPTION;
  sc = ka::TaskBeginAdaptive
    (flags, &ka::WrapperSplitter<Work<T,OP>,&Work<T,OP>::split>, &work);
  
  /* while there is sequential work to do*/
  while (work.extract_seq(beg, end))
  {
    /* apply w->op foreach item in [pos, end[ */
    std::for_each( beg, end, op );
  }
  
  /* wait for thieves */
  ka::TaskEndAdaptive(sc);
  /* here: 1/ all thieves have finish their result */

  kaapi_mem_delete_host_mappings((kaapi_mem_addr_t)base, size);
}


/**
*/
void apply_add1( double_type& v )
{
  v += 1;
}

/* My main task */
struct doit {
  void operator()(int argc, char** argv )
  {
    double t0,t1;
    double sum = 0.f;
    size_t size = 100000;
    if (argc >1) size = atoi(argv[1]);
    
    double_type* array = new double_type[size];

    for (int iter = 0; iter < 100; ++iter)
    {
      /* initialize, apply, check */
      for (size_t i = 0; i < size; ++i)
        array[i] = 0.f;
        
      t0 = kaapi_get_elapsedns();
      for_each( array, array+size, apply_add1 );
      t1 = kaapi_get_elapsedns();
      sum += (t1-t0)/1000; /* ms */

      for (size_t i = 0; i < size; ++i)
        if (array[i] != 1.f)
        {
          std::cout << "invalid @" << i << " == " << array[i] << std::endl;
          break ;
        }
    }

    std::cout << "Done " << sum/100 << " (ms)" << std::endl;
  }
};


/* main entry point : Kaapi initialization
*/
int main(int argc, char** argv)
{
  try {
    /* Join the initial group of computation : it is defining
       when launching the program by a1run.
    */
    ka::Community com = ka::System::join_community( argc, argv );
    
    /* Start computation by forking the main task */
    ka::SpawnMain<doit>()(argc, argv); 
    
    /* Leave the community: at return to this call no more athapascan
       tasks or shared could be created.
    */
    com.leave();

    /* */
    ka::System::terminate();
  }
  catch (const std::exception& E) {
    ka::logfile() << "Catch : " << E.what() << std::endl;
  }
  catch (...) {
    ka::logfile() << "Catch unknown exception: " << std::endl;
  }
  
  return 0;
}
