#include "hip/hip_runtime.h"
/*
** xkaapi
** 
** Created on Tue Mar 31 15:19:14 2009
** Copyright 2009 INRIA.
**
** Contributors :
** thierry.gautier@inrialpes.fr
** fabien.lementec@imag.fr
** 
** This software is a computer program whose purpose is to execute
** multithreaded computation with data flow synchronization between
** threads.
** 
** This software is governed by the CeCILL-C license under French law
** and abiding by the rules of distribution of free software.  You can
** use, modify and/ or redistribute the software under the terms of
** the CeCILL-C license as circulated by CEA, CNRS and INRIA at the
** following URL "http://www.cecill.info".
** 
** As a counterpart to the access to the source code and rights to
** copy, modify and redistribute granted by the license, users are
** provided only with a limited warranty and the software's author,
** the holder of the economic rights, and the successive licensors
** have only limited liability.
** 
** In this respect, the user's attention is drawn to the risks
** associated with loading, using, modifying and/or developing or
** reproducing the software by the user in light of its specific
** status of free software, that may mean that it is complicated to
** manipulate, and that also therefore means that it is reserved for
** developers and experienced professionals having in-depth computer
** knowledge. Users are therefore encouraged to load and test the
** software's suitability as regards their requirements in conditions
** enabling the security of their systems and/or data to be ensured
** and, more generally, to use and operate it in the same conditions
** as regards security.
** 
** The fact that you are presently reading this means that you have
** had knowledge of the CeCILL-C license and that you accept its
** terms.
** 
*/
#include <stdint.h>
#include <sys/types.h>
#include <hip/hip_runtime.h>
#include "kaapi++.h"
#include "for_each_work.h"

// missing decls
typedef uintptr_t kaapi_mem_addr_t;
extern "C" void kaapi_mem_delete_host_mappings(kaapi_mem_addr_t, size_t);
typedef float double_type;


/** Description of the example.

    Overview of the execution.
    
    What is shown in this example.
    
    Next example(s) to read.
*/

__global__ void add1
(double_type* array, unsigned int first, unsigned int last)
{
  const unsigned int nelems = last - first;
  const unsigned int per_thread = nelems / blockDim.x;
  unsigned int i = first + threadIdx.x * per_thread;

  unsigned int j = last;
  if (threadIdx.x != (blockDim.x - 1)) j = i + per_thread;

  for (; i < j; ++i) ++array[i];
}

/** Simple thief task that only do sequential computation
*/
template<typename T, typename OP>
struct TaskBodyCPU<TaskThief<T, OP> > {
  void operator() ( ka::pointer_rw<T> beg, ka::pointer_rw<T> end, OP op)
  {
    std::for_each( beg, end, op );
  }
};

template<typename T, typename OP>
struct TaskBodyGPU<TaskThief<T, OP> > {
  void operator()
  (ka::gpuStream stream, ka::pointer_rw<T> beg, ka::pointer_rw<T> end, OP op)
  {
    const hipStream_t custream = (hipStream_t)stream.stream;

    const size_t size = (size_t)(end - beg);

    printf("cudaTask(0x%lx 0x%lx, %lu)\n",
	   (uintptr_t)custream, beg, size);
    fflush(stdout);

    add1<<<1, 256, 0, custream>>>(beg, 0, size);
  }
};

/* For each main function */
template<typename T, class OP>
static void for_each( T* beg, T* end, OP op )
{
  T* const base = beg;
  const size_t size = (size_t)(end - beg) * sizeof(T*);

  /* range to process */
  ka::StealContext* sc;
  Work<T,OP> work(beg, end, op);

  /* push an adaptive task */
  sc = ka::TaskBeginAdaptive(
        /* flag: concurrent which means concurrence between extrac_seq & splitter executions */
          KAAPI_SC_CONCURRENT 
        /* flag: no preemption which means that not preemption will be available (few ressources) */
        | KAAPI_SC_NOPREEMPTION, 
        /* use a wrapper to specify the method to used during parallel split */
        &ka::WrapperSplitter<Work<T,OP>,&Work<T,OP>::split>,
        &work
  );
  
  /* while there is sequential work to do*/
  while (work.extract_seq(beg, end))
  {
    /* apply w->op foreach item in [pos, end[ */
    std::for_each( beg, end, op );
  }
  
  /* wait for thieves */
  ka::TaskEndAdaptive(sc);
  /* here: 1/ all thieves have finish their result */

  kaapi_mem_delete_host_mappings((kaapi_mem_addr_t)base, size);
}


/**
*/
void apply_add1( double_type& v )
{
  v += 1;
}

/* My main task */
struct doit {
  void operator()(int argc, char** argv )
  {
    double t0,t1;
    double sum = 0.f;
    size_t size = 100000;
    if (argc >1) size = atoi(argv[1]);
    
    double_type* array = new double_type[size];

    for (int iter = 0; iter < 100; ++iter)
    {
      /* initialize, apply, check */
      for (size_t i = 0; i < size; ++i)
        array[i] = 0.f;
        
      t0 = kaapi_get_elapsedns();
      for_each( array, array+size, apply_add1 );
      t1 = kaapi_get_elapsedns();
      sum += (t1-t0)/1000; /* ms */

      for (size_t i = 0; i < size; ++i)
        if (array[i] != 1.f)
        {
          std::cout << "invalid @" << i << " == " << array[i] << std::endl;
          break ;
        }
    }

    std::cout << "Done " << sum/100 << " (ms)" << std::endl;
  }
};


/* main entry point : Kaapi initialization
*/
int main(int argc, char** argv)
{
  try {
    /* Join the initial group of computation : it is defining
       when launching the program by a1run.
    */
    ka::Community com = ka::System::join_community( argc, argv );
    
    /* Start computation by forking the main task */
    ka::SpawnMain<doit>()(argc, argv); 
    
    /* Leave the community: at return to this call no more athapascan
       tasks or shared could be created.
    */
    com.leave();

    /* */
    ka::System::terminate();
  }
  catch (const ka::Exception& E) {
    ka::logfile() << "Catch : " << E.what() << std::endl;
  }
  catch (...) {
    ka::logfile() << "Catch unknown exception: " << std::endl;
  }
  
  return 0;
}
