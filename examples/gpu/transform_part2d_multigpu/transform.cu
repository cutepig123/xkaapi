#include "hip/hip_runtime.h"
#include <stdint.h>
#include <string.h>
#include <sys/types.h>
#include <hip/hip_runtime.h>
#include "kaapi++"

// missing decls
typedef uintptr_t kaapi_mem_addr_t;
extern "C" void kaapi_mem_delete_host_mappings(kaapi_mem_addr_t, size_t);
extern "C" void* kaapi_mem_alloc_host(size_t);
extern "C" void kaapi_mem_free_host(void*);
extern "C" unsigned int kaapi_cuda_get_kasid_user(size_t);
extern "C" size_t kaapi_cuda_get_proc_count(void);
extern "C" unsigned int kaapi_cuda_get_first_kid(void);

// typedefed to float since gtx280 has no double
typedef unsigned int double_type;

// static configuration
#define CONFIG_ITER_COUNT 1
#define CONFIG_ROW_COUNT 16
#define CONFIG_COL_COUNT 16
#define CONFIG_RANGE_CHECK 1

// task signature
struct TaskAddone : public ka::Task<1>::Signature
<ka::RW<ka::range2d<double_type> > > {};

// cuda kernel
__global__ void addone(double_type* array)
{
  ++array[threadIdx.y * blockDim.x + threadIdx.x];
}


// lazyness symphony

template<typename barfu>
inline static size_t rows(const barfu& o)
{ return o.dim(0); }

template<typename barfu>
inline static size_t cols(const barfu& o)
{ return o.dim(1); }


// cpu implementation

template<> struct TaskBodyCPU<TaskAddone>
{
  void operator()(ka::range2d_rw<double_type> range)
  {
    for (size_t row = 0; row < rows(range); ++row)
      for (size_t col = 0; col < cols(range); ++col)
	range(row, col) += 1;
  }

};

// gpu implementation
template<> struct TaskBodyGPU<TaskAddone>
{
  void operator()(ka::gpuStream stream, ka::range2d_rw<double_type> range)
  {
    printf(">>> kudaAddone %lx\n", (uintptr_t)&range(0, 0));

    const hipStream_t custream = (hipStream_t)stream.stream;

    static const dim3 fubar(16, 16);
    addone<<<1, fubar, 0, custream>>>(&range(0, 0));
  }
};


// init task. needed because of a bug in the runtime.
struct TaskInit : public ka::Task<1>::Signature
<ka::W<ka::range2d<double_type> > > {};

template<> struct TaskBodyCPU<TaskInit>
{
  void operator() (ka::range2d_w<double_type> range)
  {
    printf(">>> TaskInit %lx\n", (uintptr_t)&range(0, 0));

    for (size_t row = 0; row < rows(range); ++row)
      for (size_t col = 0; col < cols(range); ++col)
	range(row, col) = 0;
  }
};


// fetch memory back to original cpu
struct TaskFetch : public ka::Task<2>::Signature
<uintptr_t, ka::R<ka::range2d<double_type> > > {};

template<> struct TaskBodyCPU<TaskFetch>
{
  void operator()(uintptr_t fubar, ka::range2d_r<double_type> range)
  {
    double_type* const addr = (double_type*)fubar;

    printf(">>> TaskFetch %lx <- %lx\n", addr, (uintptr_t)&range(0, 0));

#if 0 // uncomment if valid
    const size_t size = rows(range) * cols(range) * sizeof(double_type);
    memcpy(addr, (void*)&range(0, 0), size);
#else
    for (size_t row = 0; row < rows(range); ++row)
      for (size_t col = 0; col < cols(range); ++col)
	addr[row * cols(range) + col] = range(row, col);
#endif
  }
};


/* Main of the program
*/
struct doit {
  void operator()(int argc, char** argv )
  {
    const size_t gpu_count = kaapi_cuda_get_proc_count();
    const size_t& array_count = gpu_count;

    double_type* arrays[array_count];
    const size_t total_size =
      CONFIG_ROW_COUNT * CONFIG_COL_COUNT * sizeof(double_type);

    double t0,t1, sum = 0.f;

    for (size_t iter = 0; iter < CONFIG_ITER_COUNT; ++iter)
    {
      t0 = kaapi_get_elapsedns();

      // prepare partitions
      ka::ThreadGroup threadgroup(1 + gpu_count);

      threadgroup.begin_partition();

      // set kasid users to handle multi gpus
      const unsigned int first_kid = kaapi_cuda_get_first_kid();
      const unsigned int last_kid = first_kid + gpu_count;

      for (size_t cu_part = first_kid; cu_part < last_kid; ++cu_part)
      {
	const unsigned int kasid_user =
	  kaapi_cuda_get_kasid_user(cu_part - first_kid);
	threadgroup.force_kasid(cu_part, KAAPI_PROC_TYPE_CUDA, kasid_user);
      }

      for (size_t count = 0; count < array_count; ++count)
      {
	double_type* const array = (double_type*)kaapi_mem_alloc_host(total_size);

	printf("Spawn(%lx)\n", (uintptr_t)array);

	memset(array, 0, total_size);
	ka::range2d<double_type> range
	  (array, CONFIG_ROW_COUNT, CONFIG_COL_COUNT, CONFIG_COL_COUNT);
	threadgroup.Spawn<TaskInit>(ka::SetPartition(0))(range);
	threadgroup.Spawn<TaskAddone>(ka::SetPartition(1 + count))(range);
	threadgroup.Spawn<TaskFetch>(ka::SetPartition(0))((uintptr_t)array, range);
	arrays[count] = array;
      }

      threadgroup.end_partition();

      threadgroup.execute();

      t1 = kaapi_get_elapsedns();
      sum += (t1-t0)/1000; // us

      // check it
#if CONFIG_RANGE_CHECK
      printf(">>> checking range\n");
      for (size_t count = 0; count < array_count; ++count)
      {
	double_type* const array = arrays[count];
	for (size_t row = 0; row < CONFIG_ROW_COUNT; ++row)
	  for (size_t col = 0; col < CONFIG_COL_COUNT; ++col)
	  {
	    const double_type value = array[row * CONFIG_COL_COUNT + col];
	    if (value != 1)
	    {
	      printf("invalid @%u(%u,%u) == %u\n", count, row, col, value);
	      row = CONFIG_ROW_COUNT - 1;
	      count = array_count - 1;
	      break;
	    }
	  }
      }
      printf("<<< checking range\n");
#endif

      for (size_t count = 0; count < array_count; ++count)
      {
	kaapi_mem_delete_host_mappings
	  ((kaapi_mem_addr_t)arrays[count], total_size);
	kaapi_mem_free_host(arrays[count]);
      }
    }

    printf("time: %lf (us)\n", sum / CONFIG_ITER_COUNT);
  }
};


/*
*/
int main( int argc, char** argv ) 
{
  try {
    ka::Community com = ka::System::join_community( argc, argv );

    ka::SpawnMain<doit>()(argc, argv); 

    com.leave();

    ka::System::terminate();
  }
  catch (const ka::Exception& E) {
    ka::logfile() << "Catch : "; E.print(std::cout); std::cout << std::endl;
  }
  catch (...) {
    ka::logfile() << "Catch unknown exception: " << std::endl;
  }
  return 0;    
}
