#include "hip/hip_runtime.h"
/*
 * matrix-mulitpy.cilk
 *
 * An implementation of matrix multiply based on Cilk parallelization (matrix_multiply.cilk) 
 * but using Kaapi C++ construction

 * First of five matrix multiply examples to compare dense matrix multiplication 
 * algorithms using Cilk parallelization.
 *   Example 1: Straightforward loop parallelization of matrix multiplication.
 *
 * Copyright (c) 2007-2008 Cilk Arts, Inc.  55 Cambridge Street,
 * Burlington, MA 01803.  Patents pending.  All rights reserved. You may
 * freely use the sample code to guide development of your own works,
 * provided that you reproduce this notice in any works you make that
 * use the sample code.  This sample code is provided "AS IS" without
 * warranty of any kind, either express or implied, including but not
 * limited to any implied warranty of non-infringement, merchantability
 * or fitness for a particular purpose.  In no event shall Cilk Arts,
 * Inc. be liable for any direct, indirect, special, or consequential
 * damages, or any other damages whatsoever, for any use of or reliance
 * on this sample code, including, without limitation, any lost
 * opportunity, lost profits, business interruption, loss of programs or
 * data, even if expressly advised of or otherwise aware of the
 * possibility of such damages, whether in an action of contract,
 * negligence, tort, or otherwise.
 *
 */

#include <iostream>
#include <iomanip>
#include <string>
#include <hip/hip_runtime.h>
#include "kaapi++" // this is the new C++ interface for Kaapi
#if 0
#include <cblas.h>
#endif

#if CONFIG_USE_CUBLAS
#include <hipblas.h>
#endif // CONFIG_USE_CUBLAS

#if CONFIG_USE_VOLKOV
extern void  volkov_sgemm
(hipStream_t, float*, const float*, const float*, int, int, int);
#endif

// missing definition
extern "C" int kaapi_memory_synchronize(void);

// cublas
#if CONFIG_USE_CUBLAS

// by ref values
static float alpha = 1.;
static float beta = 1.;

static hipblasHandle_t cublas_handle;

static int initialize_cublas(void) 
{
  const hipblasStatus_t status = hipblasCreate(&cublas_handle);
  if (status != HIPBLAS_STATUS_SUCCESS)
  {
    printf("hipblasCreate() == %u\n", status);
    return -1;
  }

#if 0
  hipblasSetPointerMode(cublas_handle, HIPBLAS_POINTER_MODE_DEVICE);
#else
  hipblasSetPointerMode(cublas_handle, HIPBLAS_POINTER_MODE_HOST);
#endif

  return 0;
}

static void finalize_cublas(void)
{
  hipblasDestroy(cublas_handle);
}

#endif // CONFIG_USE_CUBLAS


static int BLOCSIZE = 0;

// no double type on gtx280
typedef float double_type;


// check results
#define CONFIG_DO_CHECK 1
#if CONFIG_DO_CHECK

# include <stdlib.h>

static int do_check
(const double_type* a, const double_type* b, const double_type* c, unsigned int n)
{
  // a, b, c nxn matrices

  double_type* const tmp = (double_type*)
    malloc(n * n * sizeof(double_type));
  if (tmp == NULL) return -1;

  unsigned int i, j, k;

  for (i = 0; i < n * n; ++i) tmp[i] = 0.;

  for (i = 0; i < n; ++i)
  {
    for (j = 0; j < n; ++j)
    {
      for (k = 0; k < n; ++k)
	tmp[i * n +  j] += a[i * n + k] * b[k * n + j];
    }
  }

  int res = -1;

  for (i = 0; i < n; ++i)
  {
    for (j = 0; j < n; ++j)
    {
      k = i * n + j;
      if (abs(c[k] - tmp[k]) >= 0.001)
      {
	printf("invalid @%u,%u %f != %f\n", i, j, c[k], tmp[k]);
	goto on_error;
      }
    }
  }

  res = 0;

 on_error:
  free(tmp);

  return res;
}
#endif // CONFIG_DO_CHECK


// fetching task
struct TaskMatFetch : public ka::Task<1>::Signature<
  ka::RW<ka::range2d<double_type> > // C
>{};

template<>
struct TaskBodyCPU<TaskMatFetch> {
  void operator()( ka::range2d_rw<double_type>) {}
};


/* Task Print
 * this task prints the sum of the entries of an array 
 * each entries is view as a pointer object:
    array<1,R<int> > means that each entry may be read by the task
 */
struct TaskPrintMatrix : public ka::Task<2>::Signature<std::string,  ka::R<ka::range2d<double_type> > > {};

template<>
struct TaskBodyCPU<TaskPrintMatrix> {
  void operator() ( std::string msg, ka::range2d_r<double_type> A  )
  {
    size_t d0 = A.dim(0);
    size_t d1 = A.dim(1);

    for (size_t i = 0; i < d0; ++i)
    {
      for (size_t j = 0; j < d1; ++j)
	printf(" %.2f", A(i, j));
      printf("\n");
    }
    printf("\n");
  }
};

/**
*/
struct TaskSeqMatProduct: public ka::Task<3>::Signature<
      ka::R<ka::range2d<double_type> >, /* A */
      ka::R<ka::range2d<double_type> >,  /* B */
      ka::RW<ka::range2d<double_type> >   /* C */
>{};

template<>
struct TaskBodyCPU<TaskSeqMatProduct> {
  void operator()( ka::range2d_r<double_type> A, ka::range2d_r<double_type> B, ka::range2d_rw<double_type> C )
  {
    size_t N = A.dim(0);
    size_t M = B.dim(0);
    size_t K = C.dim(1);

#if 0    
    /* a call to blas should be more performant here */
    cblas_dgemm(
        CblasRowMajor, CblasNoTrans, CblasNoTrans,
        M, N, K, 1.0, 
        A.ptr(), A.lda(),
        B.ptr(), B.lda(),
        1.0, 
        C.ptr(), C.lda()
    );
#else
    for (size_t i =0; i<N;++i)
      for (size_t j =0; j<M; ++j)
        for (size_t k =0; k<K; ++k)
          C(i,j) += A(i,k)*B(k,j);
#endif
  }
};


// TaskSeqMatProduct gpu implementation

__global__ void mulKernel
(const double_type* a, const double_type* b, double_type* c, unsigned int m)
{
  // compute a * b = c;
  // a, b, c of size m x m
  // ldN the leading dimension

#if 1

  const unsigned int mm = m * m;

  const unsigned int per_thread = mm / blockDim.x;

  unsigned int i = threadIdx.x * per_thread;

  double_type* cpos = c + i;
  double_type* cend = cpos + per_thread;

  if (threadIdx.x == (blockDim.x - 1)) cend = c + mm;

  __syncthreads();

  // foreach c elem
  for (; cpos != cend; ++cpos, ++i)
  {
    const double_type* apos = a + (i / m) * m; // i / m rounded...
    const double_type* bpos = b + i % m;

    // ... res = innerprod(aik, bkj);
    double_type res = 0;
    for (unsigned int k = 0; k < m; ++k, ++apos, bpos += m)
      res += (*apos) * (*bpos);

    // update c
    *cpos += res;
  }

#elif 0

  if ((threadIdx.x == 0) && (threadIdx.y == 0))
  {
    for (unsigned int i = 0; i < m; ++i)
      for (unsigned int j = 0; j < m; ++j)
	for (unsigned int k = 0; k < m; ++k)
	  c[i * m + j] += a[i * m + k] * b[k * m + j];
  }

#endif
}

template<>
struct TaskBodyGPU<TaskSeqMatProduct> {
  void operator()
  (
   ka::gpuStream stream,
   ka::range2d_r<double_type> A,
   ka::range2d_r<double_type> B,
   ka::range2d_rw<double_type> C
  )
  {
    const hipStream_t custream = (hipStream_t)stream.stream;

    size_t mm = A.dim(0) * A.dim(0);
    const size_t thread_count = mm < 512 ? mm : 512;

#if CONFIG_USE_CUBLAS
    hipblasStatus_t status;

    status = hipblasSetStream(cublas_handle, custream);
    if (status != HIPBLAS_STATUS_SUCCESS)
    {
      printf("hipblasSetStream() == %u\n", status);
      return ;
    }

    const int mnk = A.dim(0);

    // warning: cublas use col major order
    // HIPBLAS_OP_N then transpose
    status = hipblasSgemm
    (
     cublas_handle,
     HIPBLAS_OP_N, HIPBLAS_OP_N,
     mnk, mnk, mnk,
     &alpha, A.ptr(), A.dim(0), B.ptr(), B.dim(0),
     &beta, C.ptr(), C.dim(0)
    );

    if (status != HIPBLAS_STATUS_SUCCESS)
    {
      printf("hipblasDgemm() == %u\n", status);
      return ;
    }
#elif CONFIG_USE_VOLKOV
    volkov_sgemm
    (
     custream,
     C.ptr(), A.ptr(), B.ptr(), A.dim(0), A.dim(1), B.dim(1)
    );
#else
    mulKernel<<<1, dim3(thread_count), 0, custream>>>
      (A.ptr(), B.ptr(), C.ptr(), A.dim(0));
#endif
  }
};

struct TaskMatProduct: public ka::Task<3>::Signature<
      ka::R<ka::range2d<double_type> >, /* A */
      ka::R<ka::range2d<double_type> >,  /* B */
      ka::RPWP<ka::range2d<double_type> >   /* C */
>{};

template<>
struct TaskBodyCPU<TaskMatProduct> {
  void operator()( ka::range2d_r<double_type> A, ka::range2d_r<double_type> B, ka::range2d_rpwp<double_type> C )
  {
    size_t M = A.dim(0);
    size_t K = B.dim(0);
    size_t N = B.dim(1);
    int bloc = BLOCSIZE;
    
    for (size_t i=0; i<M; i += bloc)
    {
      ka::rangeindex ri(i, i+bloc);
      for (size_t j=0; j<N; j += bloc)
      {
        ka::rangeindex rj(j, j+bloc);
        for (size_t k=0; k<K; k += bloc)
        {
          ka::rangeindex rk(k, k+bloc);
          ka::Spawn<TaskSeqMatProduct>()(A(ri,rk), B(rk,rj), C(ri,rj));
#if 0 // taskfetch
          ka::Spawn<TaskMatFetch>()(C(ri,rj));
#endif // taskfetch
        }
      }
    }
  }
};

/* Main of the program
*/
struct doit {
  void operator()(int ac, char** av)
  {
    // av[1] = matrix_size
    // av[2] = block_count

    const int matrix_size = atoi(av[1]);
    const int block_count = atoi(av[2]);
    BLOCSIZE = matrix_size / block_count;

    const int n = matrix_size;

    double_type* dA = (double_type*) calloc(n* n, sizeof(double_type));
    double_type* dB = (double_type*) calloc(n* n, sizeof(double_type));
    double_type* dC = (double_type*) calloc(n* n, sizeof(double_type));
    if (0 == dA || 0 == dB || 0 == dC) 
    {
        std::cout << "Fatal Error. Cannot allocate matrices A, B, and C."
            << std::endl;
        return;
    }

    for(int i = 0; i < n * n; ++i) {
      const double_type aval =
	(double_type) (((i + 1) * i) % 1024 - 512) / 512;
      const double_type bval =
	(double_type)((i * i) % 1024 - 512) / 512;

#if CONFIG_USE_VOLKOV // transpose if col major order 
      const unsigned int index = ((i / n) * n) + i % n;
#else
      const unsigned int index = i;
#endif

      dA[index] = aval;
      dB[index] = aval;
      dC[index] = 0.;
    }

    ka::array<2,double_type> A(dA, n, n, n);
    ka::array<2,double_type> B(dB, n, n, n);
    ka::array<2,double_type> C(dC, n, n, n);

#if 0 // TOREMOVE <-- running twice does not work
    ka::Spawn<TaskMatProduct>(ka::SetStaticSched())( A, B, C );
    ka::Sync();
#endif // TOREMOVE

    // Multiply to get C = A*B 
    double t0 = kaapi_get_elapsedtime();
    ka::Spawn<TaskMatProduct>(ka::SetStaticSched())( A, B, C );
    ka::Sync();

    // dont time memory sync for the benchmarks since
    // it does not reflect the execution pipeline
    double t1 = kaapi_get_elapsedtime();

    kaapi_memory_synchronize();

    std::cout << t1 - t0; // seconds

    // If n is small, print the results
#if 0
    ka::Spawn<TaskPrintMatrix>()( std::string("C"), C );
    ka::Sync();
#endif

#if CONFIG_DO_CHECK
    if (do_check(dA, dB, dC, n) == -1)
      printf("invalid matrix\n");
#endif

    free(dA);
    free(dB);
    free(dC);
  }
};


/* main entry point : Kaapi initialization
*/
int main(int argc, char** argv)
{
#if CONFIG_USE_CUBLAS
  if (initialize_cublas() == -1) return ;
#endif

  try {
    /* Join the initial group of computation : it is defining
       when launching the program by a1run.
    */
    ka::Community com = ka::System::join_community( argc, argv );
    
    /* Start computation by forking the main task */
    ka::SpawnMain<doit>()(argc, argv); 
    
    /* Leave the community: at return to this call no more athapascan
       tasks or shared could be created.
    */
    com.leave();

    /* */
    ka::System::terminate();
  }
  catch (const std::exception& E) {
    ka::logfile() << "Catch : " << E.what() << std::endl;
  }
  catch (...) {
    ka::logfile() << "Catch unknown exception: " << std::endl;
  }

#if CONFIG_USE_CUBLAS
  finalize_cublas();
#endif
  
  return 0;
}

